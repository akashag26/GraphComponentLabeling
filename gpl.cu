#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#define MAX_THREADS_PER_BLOCK 512 

int no_of_nodes;
int edge_list_size;
FILE *fp;

//Structure to hold a node information
struct Node
{
	int starting;
	int no_of_edges;
};

__global__ void 
Kernel3(Node* g_graph_nodes, int* g_graph_edges,int* cd, bool* f1d, bool *f2d,int no_of_nodes,bool *md)
{
        int i = blockIdx.x*MAX_THREADS_PER_BLOCK + threadIdx.x;
	//int idx, idxi;
	//printf("\n Thread id:  %d",i);
	int  ci, cj; 	
	if(f1d[i]==true)
	{
		f1d[i]=false;
	
		ci=cd[i];
		bool cimod=false;
		int temp;
for(int j=g_graph_nodes[i].starting;j<(g_graph_nodes[i].starting + g_graph_nodes[i].no_of_edges); j++)
		{
			temp=g_graph_edges[j];
			cj = cd[temp];
			if ( ci < cj )
			{
				atomicMin(&cd[temp],ci);
				f2d[temp]=true;
				*md=true;				
			}
			else
			if (ci>cj)
			{
				ci=cj;
				cimod=true;
			}	
		}
		
		if(cimod==true)
		{
			atomicMin(&cd[i],ci);
			f2d[i]=true;
			*md=true;
		}
	
	}
	//	printf("\n End of kernel:  %d", cd[i]);
}


long long start_timer();
long long stop_timer(long long start_time, char *name);


void GPLGraph(int argc, char** argv);

////////////////////////////////////////////////////////////////////////////////
// Main Program
////////////////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
	no_of_nodes=0;
	edge_list_size=0;
	GPLGraph( argc, argv);
}

void Usage(int argc, char**argv){

fprintf(stderr,"Usage: %s <input_file>\n", argv[0]);

}
////////////////////////////////////////////////////////////////////////////////
////////////////////////////////////////////////////////////////////////////////
void GPLGraph( int argc, char** argv) 
{

    char *input_f;
	if(argc!=2){
	Usage(argc, argv);
	exit(0);
	}
	
	input_f = argv[1];
	printf("Reading File\n");
	//Read in Graph from a file
	fp = fopen(input_f,"r");
	if(!fp)
	{
		printf("Error Reading graph file\n");
		return;
	}

	int source = 0;

	fscanf(fp,"%d",&no_of_nodes);

	int num_of_blocks = 1;
	int num_of_threads_per_block = no_of_nodes;

	//Make execution Parameters according to the number of nodes
	//Distribute threads across multiple Blocks if necessary
	if(no_of_nodes>MAX_THREADS_PER_BLOCK)
	{
		num_of_blocks = (int)ceil(no_of_nodes/(double)MAX_THREADS_PER_BLOCK); 
		num_of_threads_per_block = MAX_THREADS_PER_BLOCK; 
	}

	// allocate host memory
	Node* h_graph_nodes = (Node*) malloc(sizeof(Node)*no_of_nodes);

	int start, edgeno;   
	// initalize the memory
	for( unsigned int i = 0; i < no_of_nodes; i++) 
	{
		fscanf(fp,"%d %d",&start,&edgeno);
		h_graph_nodes[i].starting = start;
		h_graph_nodes[i].no_of_edges = edgeno;
	}

	//read the source node from the file
	fscanf(fp,"%d",&source);
	source=0;


	fscanf(fp,"%d",&edge_list_size);

	int id,cost;
	int* h_graph_edges = (int*) malloc(sizeof(int)*edge_list_size);
	for(unsigned int i=0; i < edge_list_size ; i++)
	{
		fscanf(fp,"%d",&id);
		fscanf(fp,"%d",&cost);
		h_graph_edges[i] = id;
	}

	  int* c = (int*) malloc(sizeof(int)*no_of_nodes);
	 bool* f1 = (bool*) malloc(sizeof(bool)*no_of_nodes);
 	bool* f2 = (bool*) malloc(sizeof(bool)*no_of_nodes);       
	bool* f3 = (bool*) malloc(sizeof(bool)*no_of_nodes);


 for(unsigned int i=0; i < no_of_nodes ; i++)
        {
        	c[i]=i;
                f1[i]=true;
                f2[i]=false;
        }
	
	if(fp)
		fclose(fp);    

	printf("Read File\n");

	//Copy the Node list to device memory
	Node* d_graph_nodes;
	hipMalloc( (void**) &d_graph_nodes, sizeof(Node)*no_of_nodes) ;
	hipMemcpy( d_graph_nodes, h_graph_nodes, sizeof(Node)*no_of_nodes, hipMemcpyHostToDevice) ;

	//Copy the Edge List to device Memory
	int* d_graph_edges;
	hipMalloc( (void**) &d_graph_edges, sizeof(int)*edge_list_size) ;
	hipMemcpy( d_graph_edges, h_graph_edges, sizeof(int)*edge_list_size, hipMemcpyHostToDevice) ;

	//Allocate Color Array in device Memory
	int* cd;
	hipMalloc( (void**) &cd, sizeof(int)*no_of_nodes);
	hipMemcpy( cd, c, sizeof(int)*no_of_nodes,hipMemcpyHostToDevice);

	//Allocate Boolean Array in current Iteration
	bool* f1d;
	hipMalloc( (void**) &f1d, sizeof(bool)*no_of_nodes);
	hipMemcpy( f1d, f1, sizeof(bool)*no_of_nodes,hipMemcpyHostToDevice);

	//Allocate Boolean Array for next Iteration
	bool* f2d;
	hipMalloc( (void**) &f2d, sizeof(bool)*no_of_nodes);
	hipMemcpy( f2d, f2, sizeof(bool)*no_of_nodes,hipMemcpyHostToDevice);

 bool* f3d;
        hipMalloc( (void**) &f3d, sizeof(bool)*no_of_nodes);
       
	
	bool m;
	bool *md;
	hipMalloc( (void**) &md, sizeof(bool));


	printf("Copied Everything to Kernel");

	// setup execution parameters
	dim3  grid( num_of_blocks, 1, 1);
	dim3  threads( num_of_threads_per_block, 1, 1);
long long timer;
	timer = start_timer();
	int k=0;
	printf("Start traversing the tree\n");
	
	//Call the Kernel untill all the elements of Frontier are not false
	do
	{
		m=false;
		//if no thread changes this value then the loop stops
		hipMemcpy( md, &m, sizeof(bool), hipMemcpyHostToDevice) ;
		
Kernel3<<< grid, threads, 0 >>>( d_graph_nodes, d_graph_edges,cd,f1d,f2d, no_of_nodes,md);
		// check if kernel execution generated and error
		
	hipMemcpy( f1, f1d, sizeof(bool)*no_of_nodes,hipMemcpyDeviceToHost);
	hipMemcpy( f2, f2d, sizeof(bool)*no_of_nodes,hipMemcpyDeviceToHost);
	hipMemcpy( f1d, f2, sizeof(bool)*no_of_nodes,hipMemcpyHostToDevice);
 	hipMemcpy( f2d, f1, sizeof(bool)*no_of_nodes,hipMemcpyHostToDevice);
	k++;
	hipMemcpy( &m,md , sizeof(bool), hipMemcpyDeviceToHost) ;
//		printf("\n \n Return from kernel:   %d",m);
	}
	while(m);


	printf("Kernel Executed %d times\n",k);

	// copy result from device to host
	hipMemcpy( c,cd, sizeof(int)*no_of_nodes, hipMemcpyDeviceToHost) ;

	//Store the result into a file
	FILE *fpo = fopen("result.txt","w");
	for(unsigned int i=0;i<no_of_nodes;i++)
		fprintf(fpo,"%d) color:%d\n",i,c[i]);
	fclose(fpo);
	printf("Result stored in result.txt\n");


	// cleanup memory
	free( h_graph_nodes);
	free( h_graph_edges);
	hipFree(d_graph_nodes);
	hipFree(d_graph_edges);
	hipFree(cd);
	hipFree(f1d);
	hipFree(f2d);
	//cudaFree(md);
stop_timer(timer, "Total Processing time");



}
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}

long long stop_timer(long long start_time, char *label) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
	printf("%s: %.5f sec\n", label, ((float) (end_time - start_time)) / (1000 * 1000));
	return end_time - start_time;
}

